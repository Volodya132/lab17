#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>
#include <Windows.h>
#include <iomanip>

using namespace std;

__global__ void compute(double* a, double* b, double* c, double* d, double* res)
{
    int idx = threadIdx.x;
    res[idx] = (a[idx] + b[idx] + c[idx] - d[idx])/2.;
}

void randomGenerateArr(double* arr, int size) {
    for (int i = 0; i < size; i++) {
        arr[i] = rand() % 100;
    }
}

void printArr(double* arr, int size) {
    cout << fixed;
    cout.precision(2);

    for (int i = 0; i < size; i++) {
        cout << arr[i] << " ";
    }
    cout << endl;
}
int main()
{
    SetConsoleCP(1251);
    SetConsoleOutputCP(1251);

    srand(time(NULL));
    const int size = 7;
    double cpu_arr1[size];
    double cpu_arr2[size];
    double cpu_arr3[size];
    double cpu_arr4[size];
    double cpu_res[size];

    double* gpu_arr1, * gpu_arr2, * gpu_arr3, *gpu_arr4, *gpu_res;
    randomGenerateArr(cpu_arr1, size);
    randomGenerateArr(cpu_arr2, size);
    randomGenerateArr(cpu_arr3, size);
    randomGenerateArr(cpu_arr4, size);

    cout << "a: " << endl;
    printArr(cpu_arr1, size);

    cout << "b: " << endl;
    printArr(cpu_arr2, size);

    cout << "c: " << endl;
    printArr(cpu_arr3, size);

    cout << "d: " << endl;
    printArr(cpu_arr4, size);

    hipMalloc((void**)&gpu_arr1, sizeof(double)*size);
    hipMalloc((void**)&gpu_arr2, sizeof(double) * size);
    hipMalloc((void**)&gpu_arr3, sizeof(double) * size);
    hipMalloc((void**)&gpu_arr4, sizeof(double) * size);
    hipMalloc((void**)&gpu_res, sizeof(double) * size);

    hipMemcpy(gpu_arr1, cpu_arr1, sizeof(double) * size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_arr2, cpu_arr2, sizeof(double) * size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_arr3, cpu_arr3, sizeof(double) * size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_arr4, cpu_arr4, sizeof(double) * size, hipMemcpyHostToDevice);

    compute << <1, size >> > (gpu_arr1, gpu_arr2, gpu_arr3, gpu_arr4, gpu_res);

    hipMemcpy(cpu_res, gpu_res, sizeof(double) * size, hipMemcpyDeviceToHost);
    
    cout << "res: " << endl;
    printArr(cpu_res, size);


    hipFree(gpu_arr1);
    hipFree(gpu_arr2);
    hipFree(gpu_arr3);
    hipFree(gpu_arr4);
    hipFree(gpu_res);



    return 0;
}

